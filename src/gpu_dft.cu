#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "complex.h"
#include "mdscatter.h"

__constant__ float rsq;


//const size_t N_PTS = 6144; // 2048 x 3 
const int N_PTS = 512;

void __global__ gpuDFT(unsigned npts, float * pts, unsigned nq, float * qvals, cucomplex_t * ft) {

    const cucomplex_t NEG_I = make_hipFloatComplex(0.f, -1.f);

	int idx = blockDim.x * blockIdx.x  + threadIdx.x;

    __shared__ float shamem [N_PTS * 3];
	int nfills =  npts / N_PTS;
    if (npts % N_PTS) nfills++;

    int np = N_PTS;
    for (int ifill = 0; ifill < nfills; ifill++) {
        int pnt_idx = ifill * N_PTS + threadIdx.x;
        if ((ifill + 1) * N_PTS > npts) np = npts % N_PTS; 

        if (pnt_idx < npts ) {
            for (int k = 0; k < 3; k++) 
                shamem[threadIdx.x * 3 + k] = pts[pnt_idx * 3 + k];
        }

            // compute dft
        if (idx < nq) {
            for (unsigned j = 0; j < np; j++) {
			    float q_r = 0;
			    for (unsigned k = 0; k < 3; k++) 
				    q_r += qvals[3 * idx + k] * shamem[3 * j + k];
			    ft[idx] = ft[idx] + Cexpf(NEG_I * q_r); 
		    }
	    }
    }
}

py::array gpu_dft(np_array_t Pts, np_array_t qVals) {

    if ((Pts.ndim() != 2) or qVals.ndim() != 2)
        throw std::runtime_error("Input arrays must be 2-D numpy arrays");

    if ((Pts.shape()[1] != 3) or (qVals.shape()[1] != 3))
        throw std::runtime_error("input arrays must of shape [N, 3]");

    /* size of input arrays */
    unsigned npts = Pts.shape()[0];
    unsigned nq = qVals.shape()[0];

    /* NumPy  will allocate the buffer */
    auto result = py::array_t<complex_t>(nq);

    float * pts = (float *) Pts.request().ptr;
    float * qvals = (float *) qVals.request().ptr;
    complex_t * ft = (complex_t *) result.request().ptr;
	
    // copy beam-radius (squared) to constant memory
    //float brsq = beam_radius * beam_radius;
    //hipMemcpyToSymbol(HIP_SYMBOL(rsq), &brsq, sizeof(float), 0, hipMemcpyHostToDevice);

	// allocate memory on device
	float * dpts, * dqvals;
	hipMalloc((void **) &dpts, sizeof(float) * npts * 3);
	hipMalloc((void **) &dqvals, sizeof(float) * nq * 3);

    // copy arrays to device memory
	hipMemcpy(dpts, pts, sizeof(float) * 3 * npts, hipMemcpyHostToDevice);
	hipMemcpy(dqvals, qvals, sizeof(float) * 3 * nq, hipMemcpyHostToDevice);

	// allocate memory for output
	cucomplex_t * dft = NULL;
	hipMalloc((void **) &dft, sizeof(cucomplex_t) * nq);
    hipMemset(dft, sizeof(cucomplex_t) * nq, 0);

	// device parameters
	unsigned threads = N_PTS;
	unsigned blocks = nq / threads; 
	if (nq % threads != 0) blocks++;
	gpuDFT<<< blocks, threads >>> (npts, dpts, nq, dqvals, dft);

	// copy results back to host
	hipMemcpy(ft, dft, sizeof(complex_t) * nq, hipMemcpyDeviceToHost);

	// free memory
	hipFree(dpts);
	hipFree(dqvals);
	hipFree(dft);
    return result;
}
