#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "complex.h"

void __global__ gpuDFT(unsigned npts, float * pts, unsigned nq, float * qvals, cucomplex_t * ft) {
	/* lucky we have excatly 4000 particles. 4000 X 3 should fit on most modern GPUs.
     * If more particles are needed in future, we need to do some memory maneuvering gymnastics.
	 */
	const cucomplex_t NEG_I = make_hipFloatComplex(0.f, -1.f);
	// compute dft
	unsigned i = blockDim.x * blockIdx.x  + threadIdx.x;
	if (i < nq) {
		ft[i] = make_hipFloatComplex(0.f, 0.f);
		for (unsigned j = 0; j < npts; j++) {
			float q_r = 0;
			for (unsigned k = 0; k < 3; k++) 
				q_r += qvals[3 * i + k] * pts[3 * j + k];
			ft[i] = ft[i] + Cexpf(NEG_I * q_r);
		}
	}
}

void cudft(unsigned npts, float * pts, unsigned nq, float * qvals,
			complex_t * output) {

	// allocate memory on device
	float * dpts, * dqvals;
	hipMalloc((void **) &dpts, sizeof(float) * npts * 3);
	hipMalloc((void **) &dqvals, sizeof(float) * nq * 3);

    // copy arrays to device memory
	hipMemcpy(dpts, pts, sizeof(float) * 3 * npts, hipMemcpyHostToDevice);
	hipMemcpy(dqvals, qvals, sizeof(float) * 3 * nq, hipMemcpyHostToDevice);

	// allocate memory for output
	cucomplex_t * dft = NULL;
	hipMalloc((void **) &dft, sizeof(cucomplex_t) * nq);

	// device parameters
	unsigned threads = 256;
	unsigned blocks = nq / threads; 
	if (nq % threads != 0) blocks++;
	gpuDFT<<< blocks, threads >>> (npts, dpts, nq, dqvals, dft);

	// copy results back to host
	hipMemcpy(output, dft, sizeof(complex_t) * nq, hipMemcpyDeviceToHost);

	// free memory
	hipFree(dpts);
	hipFree(dqvals);
	hipFree(dft);
}
