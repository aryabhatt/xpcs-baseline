#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "complex.h"

__constant__ float rsq;

//const size_t N_PTS = 6144; // 2048 x 3 

void __global__ gpuDFT(unsigned npts, float * pts, unsigned nq, float * qvals, cucomplex_t * ft) {
	 
	const cucomplex_t NEG_I = make_hipFloatComplex(0.f, -1.f);
	// compute dft

	unsigned i = blockDim.x * blockIdx.x  + threadIdx.x;

	if (i < nq) {
		ft[i] = make_hipFloatComplex(0.f, 0.f);
		for (unsigned j = 0; j < npts; j++) {
			float q_dot_r = 0;
			for (unsigned k = 0; k < 3; k++) 
				q_dot_r += qvals[3 * i + k] * pts[3 * j + k];
			ft[i] = ft[i] + Cexpf(NEG_I * q_dot_r);
		}
	}
}

void cudft(unsigned npts, float * pts, unsigned nq, float * qvals,
			complex_t * output) {

	// allocate memory on device
	float * dpts, * dqvals;
	hipMalloc((void **) &dpts, sizeof(float) * npts * 3);
	hipMalloc((void **) &dqvals, sizeof(float) * nq * 3);

    // copy arrays to device memory
	hipMemcpy(dpts, pts, sizeof(float) * 3 * npts, hipMemcpyHostToDevice);
	hipMemcpy(dqvals, qvals, sizeof(float) * 3 * nq, hipMemcpyHostToDevice);

	// allocate memory for output
	cucomplex_t * dft = NULL;
	hipMalloc((void **) &dft, sizeof(cucomplex_t) * nq);

	// device parameters
	unsigned threads = 256;
	unsigned blocks = nq / threads; 
	if (nq % threads != 0) blocks++;
	gpuDFT<<< blocks, threads >>> (npts, dpts, nq, dqvals, dft);

	// copy results back to host
	hipMemcpy(output, dft, sizeof(complex_t) * nq, hipMemcpyDeviceToHost);

	// free memory
	hipFree(dpts);
	hipFree(dqvals);
	hipFree(dft);
}
